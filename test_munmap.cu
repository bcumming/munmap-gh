
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <cstring>
#include <sstream>
#include <iostream>
#include <new>
#include <sys/mman.h>
#include <chrono>

#include <unistd.h>
long page_size = sysconf (_SC_PAGESIZE);

void benchmark(int iterations, size_t sz, bool use_mmap, bool touch_memory) {
  std::stringstream ss;
  ss << (use_mmap ? "mmap" : "new") << "_" << (touch_memory ? "touch" : "no_touch") << "_";
  std::string testcase_name = ss.str();

  double total_alloc_time_ms = 0;
  double total_release_time_ms = 0;

  for(int i = 0; i < iterations; i++) {
    char *p;

    auto alloc_start = std::chrono::steady_clock::now();
    if (use_mmap) {
      p = (char*) mmap(NULL, sz+i*100000, PROT_READ | PROT_WRITE, MAP_ANONYMOUS | MAP_SHARED, -1, 0);
    } else {
      p = new char[sz];
    }
    auto alloc_end = std::chrono::steady_clock::now();
    std::chrono::duration<double, std::milli> alloc_duration = alloc_end - alloc_start;
    total_alloc_time_ms += alloc_duration.count();

    if (p == nullptr) std::abort();

    if (p == MAP_FAILED) {
      std::cerr << "Map failed: " << strerror(errno) << std::endl;
      std::abort();
    }

    if (touch_memory) {
      for (int j = 0; j < sz; j += page_size) {
        p[j] = 42;
      }
    }

    auto release_start = std::chrono::steady_clock::now();
    if (use_mmap) {
      if (munmap(p, sz) < 0) std::abort();
    } else {
      delete[] p;
    }

    // C timers have easier syntax...
    auto release_end = std::chrono::steady_clock::now();
    std::chrono::duration<double, std::milli> release_duration = release_end - release_start;
    total_release_time_ms += release_duration.count();
  }

  std::cout << "&&&& PERF " << testcase_name << "alloc " << total_alloc_time_ms / iterations << std::endl;
  std::cout << "&&&& PERF " << testcase_name << "release " << total_release_time_ms / iterations << std::endl;
}


int main(int argc, char* argv[]) {
  if (argc < 3) {
    std::cerr << "Takes two argument for #TLB shootdowns and alloc size" << std::endl;
    std::abort();
  }

  int iterations = atoi(argv[1]);
  std::cout << "#TLB shootdowns: " << iterations << " with page size " << page_size << std::endl;
  assert(iterations > 0);
  size_t sz = (size_t)atof(argv[2]);
  assert(sz > 0);

  if (hipSuccess != hipFree(0)) {
    std::cerr << "cudaFree(0) failed\n";
    std::abort();
  }

  void* ptr;
  hipMalloc(&ptr, 70'000'000'000);

  benchmark(iterations, sz, false, false);
  //benchmark(iterations, sz, false, true);
  benchmark(iterations, sz, true, false);
  //benchmark(iterations, sz, true, true);

  return 0;
}

